// 20-09-10

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cassert>
#include <iostream>
using std::cout;
using std::endl;

#include "format.hh"
#include "gather_scatter.cuh"

template <typename DType>
void cuSZ::impl::new_gather(
    DType*    d_A,  //
    size_t    len,
    const int m,
    int*      nnz,
    int**     csrRowPtr,
    int**     csrColInd,
    DType**   csrVal)
{
    hipsparseHandle_t   handle      = nullptr;
    hipStream_t       stream      = nullptr;
    hipsparseMatDescr_t descr       = nullptr;
    hipsparseStatus_t   status      = HIPSPARSE_STATUS_SUCCESS;
    hipError_t        cudaStat1   = hipSuccess;
    hipError_t        cudaStat2   = hipSuccess;
    hipError_t        cudaStat3   = hipSuccess;
    const int          lda         = m;
    const int          n           = m;
    int*               d_csrRowPtr = nullptr;
    int*               d_csrColInd = nullptr;
    DType*             d_csrVal    = nullptr;

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);  // 1. create stream
    assert(hipSuccess == cudaStat1);                                       //
    status = hipsparseCreate(&handle);                                       // 2. create handle
    assert(HIPSPARSE_STATUS_SUCCESS == status);                              //
    status = hipsparseSetStream(handle, stream);                             // 3. bind stream
    assert(HIPSPARSE_STATUS_SUCCESS == status);                              //
    status = hipsparseCreateMatDescr(&descr);                                // 4. create descr
    assert(HIPSPARSE_STATUS_SUCCESS == status);                              //
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);               // zero based
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);                // type

    // query workspace
    // clang-format off
    cudaStat1 = hipMalloc((void**)&d_csrRowPtr, sizeof(int)   * (m + 1));
    cudaStat2 = hipMalloc((void**)&d_csrColInd, sizeof(int)   * *nnz   );
    cudaStat3 = hipMalloc((void**)&d_csrVal,    sizeof(DType) * *nnz   );
    // clang-format on
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    // compute nnz
    int* d_nnzPerRow = nullptr;
    status           = hipsparseSnnz(
        handle, HIPSPARSE_DIRECTION_ROW,  // parsed by row
        m, n, descr, d_A, lda,           // descrption of d_A
        d_nnzPerRow, nnz);               // output
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    // step 5: dense to csr
    status = hipsparseSdense2csr(
        handle,                               //
        m, n, descr, d_A, lda,                // descritpion of d_A
        d_nnzPerRow,                          // prefileld by nnz() func
        d_csrVal, d_csrRowPtr, d_csrColInd);  // output
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    // clang-format off
    cudaStat1 = hipMemcpy(*csrRowPtr, d_csrRowPtr, sizeof(int)   * (m + 1), hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(*csrColInd, d_csrColInd, sizeof(int)   * *nnz,    hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(*csrVal,    d_csrVal,    sizeof(DType) * *nnz,    hipMemcpyDeviceToHost);
    // clang-format on
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    // clean up
    if (d_csrRowPtr) hipFree(d_csrRowPtr);
    if (d_csrColInd) hipFree(d_csrColInd);
    if (d_csrVal) hipFree(d_csrVal);
    if (d_nnzPerRow) hipFree(d_nnzPerRow);

    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (descr) hipsparseDestroyMatDescr(descr);
}

template void cuSZ::impl::new_gather<float>(float*, size_t, const int, int*, int**, int**, float**);

template <typename DType>
void cuSZ::impl::new_scatter(
    DType*    d_A,  //
    size_t    len,
    const int m,
    int*      nnz,
    int**     csrRowPtr,
    int**     csrColInd,
    DType**   csrVal)
{
    hipsparseHandle_t   handle      = nullptr;
    hipStream_t       stream      = nullptr;
    hipsparseMatDescr_t descr       = nullptr;
    hipsparseStatus_t   status      = HIPSPARSE_STATUS_SUCCESS;
    hipError_t        cudaStat1   = hipSuccess;
    hipError_t        cudaStat2   = hipSuccess;
    hipError_t        cudaStat3   = hipSuccess;
    const int          lda         = m;
    const int          n           = m;
    int*               d_csrRowPtr = nullptr;
    int*               d_csrColInd = nullptr;
    DType*             d_csrVal    = nullptr;

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);  // 1. create stream
    assert(hipSuccess == cudaStat1);                                       //
    status = hipsparseCreate(&handle);                                       // 2. create handle
    assert(HIPSPARSE_STATUS_SUCCESS == status);                              //
    status = hipsparseSetStream(handle, stream);                             // 3. bind stream
    assert(HIPSPARSE_STATUS_SUCCESS == status);                              //
    status = hipsparseCreateMatDescr(&descr);                                // 4. create descr
    assert(HIPSPARSE_STATUS_SUCCESS == status);                              //
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);               //
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);                //

    // set space
    // clang-format off
    cudaStat1 = hipMemcpy(d_csrRowPtr, *csrRowPtr, sizeof(int)   * (m + 1), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_csrColInd, *csrColInd, sizeof(int)   * *nnz,    hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(d_csrVal,    *csrVal,    sizeof(DType) * *nnz,    hipMemcpyHostToDevice);
    // clang-format on
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    // fill
    status = hipsparseScsr2dense(handle, m, n, descr, d_csrVal, d_csrRowPtr, d_csrColInd, d_A, lda);
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    if (d_csrRowPtr) hipFree(d_csrRowPtr);
    if (d_csrColInd) hipFree(d_csrColInd);
    if (d_csrVal) hipFree(d_csrVal);

    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (descr) hipsparseDestroyMatDescr(descr);
}

template void cuSZ::impl::new_scatter<float>(float*, size_t, const int, int*, int**, int**, float**);

void cuSZ::impl::GatherOutlierUsingCusparse(
    float*    d_A,  //
    size_t    len,
    const int m,
    int&      nnzC,
    int**     csrRowPtrC,
    int**     csrColIndC,
    float**   csrValC)
{
    hipsparseHandle_t   handle    = nullptr;
    hipStream_t       stream    = nullptr;
    hipsparseMatDescr_t descrC    = nullptr;
    hipsparseStatus_t   status    = HIPSPARSE_STATUS_SUCCESS;
    hipError_t        cudaStat1 = hipSuccess;
    hipError_t        cudaStat2 = hipSuccess;
    hipError_t        cudaStat3 = hipSuccess;
    // hipError_t cudaStat4 = hipSuccess;
    // hipError_t cudaStat5 = hipSuccess;
    // const int m           = 1;
    // const int n           = len;
    const int lda = m;
    const int n   = m;  // square

    // int*   csrRowPtrC = nullptr;
    // int*   csrColIndC = nullptr;
    // float* csrValC    = nullptr;
    // float* d_A        = nullptr;
    int*   d_csrRowPtrC = nullptr;
    int*   d_csrColIndC = nullptr;
    float* d_csrValC    = nullptr;

    size_t lworkInBytes = 0;
    char*  d_work       = nullptr;

    //    int nnzC = 0;

    float threshold = 0; /* remove Aij <= 4.1 */

    /* step 1: create cusparse handle, bind a stream */
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsparseCreate(&handle);
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    status = hipsparseSetStream(handle, stream);
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    /* step 2: configuration of matrix C */
    status = hipsparseCreateMatDescr(&descrC);
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);

    //    cudaStat1 = hipMalloc((void**)&d_A, sizeof(float) * lda * n);
    cudaStat2 = hipMalloc((void**)&d_csrRowPtrC, sizeof(int) * (m + 1));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    /* step 3: query workspace */
    //    cudaStat1 = hipMemcpy(d_A, A, sizeof(float) * lda * n, hipMemcpyHostToDevice);
    //    assert(hipSuccess == cudaStat1);

    status = hipsparseSpruneDense2csr_bufferSizeExt(  //
        handle,                                      //
        m,                                           //
        n,                                           //
        d_A,                                         //
        lda,                                         //
        &threshold,                                  //
        descrC,                                      //
        d_csrValC,                                   //
        d_csrRowPtrC,                                //
        d_csrColIndC,                                //
        &lworkInBytes);
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    //    printf("lworkInBytes (prune) = %lld \n", (long long)lworkInBytes);

    if (nullptr != d_work) {
        hipFree(d_work);
    }
    cudaStat1 = hipMalloc((void**)&d_work, lworkInBytes);
    assert(hipSuccess == cudaStat1);

    /* step 4: compute csrRowPtrC and nnzC */
    status = hipsparseSpruneDense2csrNnz(  //
        handle,                           //
        m,                                //
        n,                                //
        d_A,                              //
        lda,                              //
        &threshold,                       //
        descrC,                           //
        d_csrRowPtrC,                     //
        &nnzC,                            // host
        d_work);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    cudaStat1 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);

    if (0 == nnzC) cout << log_info << "No outlier." << endl;

    /* step 5: compute csrColIndC and csrValC */
    cudaStat1 = hipMalloc((void**)&d_csrColIndC, sizeof(int) * nnzC);
    cudaStat2 = hipMalloc((void**)&d_csrValC, sizeof(float) * nnzC);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    status = hipsparseSpruneDense2csr(  //
        handle,                        //
        m,                             //
        n,                             //
        d_A,                           //
        lda,                           //
        &threshold,                    //
        descrC,                        //
        d_csrValC,                     //
        d_csrRowPtrC,                  //
        d_csrColIndC,                  //
        d_work);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    cudaStat1 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);

    /* step 6: output C */
    //    csrRowPtrC = (int*)malloc(sizeof(int) * (m + 1));
    //    csrColIndC = (int*)malloc(sizeof(int) * nnzC);
    //    csrValC    = (float*)malloc(sizeof(float) * nnzC);
    *csrRowPtrC = new int[m + 1];
    *csrColIndC = new int[nnzC];
    *csrValC    = new float[nnzC];
    assert(nullptr != csrRowPtrC);
    assert(nullptr != csrColIndC);
    assert(nullptr != csrValC);

    cudaStat1 = hipMemcpy(*csrRowPtrC, d_csrRowPtrC, sizeof(int) * (m + 1), hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(*csrColIndC, d_csrColIndC, sizeof(int) * nnzC, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(*csrValC, d_csrValC, sizeof(float) * nnzC, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    //    printCsr(m, n, nnzC, descrC, csrValC, csrRowPtrC, csrColIndC, "C");

    /* free resources */
    if (d_A) hipFree(d_A);
    if (d_csrRowPtrC) hipFree(d_csrRowPtrC);
    if (d_csrColIndC) hipFree(d_csrColIndC);
    if (d_csrValC) hipFree(d_csrValC);

    //    if (csrRowPtrC) free(csrRowPtrC);
    //    if (csrColIndC) free(csrColIndC);
    //    if (csrValC) free(csrValC);

    //    for (auto i = 0; i < 200; i++) cout << i << "\t" << csrColIndC[i] << "\t" << csrValC[i] << endl;

    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (descrC) hipsparseDestroyMatDescr(descrC);

    //    hipDeviceReset();
}

void cuSZ::impl::GatherOutlierUsingCusparse(
    float*  d_A,  //
    size_t  len,
    int&    nnzC,
    int**   csrRowPtrC,
    int**   csrColIndC,
    float** csrValC)
{
    hipsparseHandle_t   handle    = nullptr;
    hipStream_t       stream    = nullptr;
    hipsparseMatDescr_t descrC    = nullptr;
    hipsparseStatus_t   status    = HIPSPARSE_STATUS_SUCCESS;
    hipError_t        cudaStat1 = hipSuccess;
    hipError_t        cudaStat2 = hipSuccess;
    hipError_t        cudaStat3 = hipSuccess;
    //    hipError_t        cudaStat4 = hipSuccess;
    //    hipError_t        cudaStat5 = hipSuccess;
    const int m   = 1;
    const int n   = len;
    const int lda = m;

    //    int*   csrRowPtrC = nullptr;
    //    int*   csrColIndC = nullptr;
    //    float* csrValC    = nullptr;

    //    float* d_A          = nullptr;
    int*   d_csrRowPtrC = nullptr;
    int*   d_csrColIndC = nullptr;
    float* d_csrValC    = nullptr;

    size_t lworkInBytes = 0;
    char*  d_work       = nullptr;

    //    int nnzC = 0;

    float threshold = 0; /* remove Aij <= 4.1 */

    /* step 1: create cusparse handle, bind a stream */
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsparseCreate(&handle);
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    status = hipsparseSetStream(handle, stream);
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    /* step 2: configuration of matrix C */
    status = hipsparseCreateMatDescr(&descrC);
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);

    //    cudaStat1 = hipMalloc((void**)&d_A, sizeof(float) * lda * n);
    cudaStat2 = hipMalloc((void**)&d_csrRowPtrC, sizeof(int) * (m + 1));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    /* step 3: query workspace */
    //    cudaStat1 = hipMemcpy(d_A, A, sizeof(float) * lda * n, hipMemcpyHostToDevice);
    //    assert(hipSuccess == cudaStat1);

    status = hipsparseSpruneDense2csr_bufferSizeExt(  //
        handle,                                      //
        m,                                           //
        n,                                           //
        d_A,                                         //
        lda,                                         //
        &threshold,                                  //
        descrC,                                      //
        d_csrValC,                                   //
        d_csrRowPtrC,                                //
        d_csrColIndC,                                //
        &lworkInBytes);
    assert(HIPSPARSE_STATUS_SUCCESS == status);

    //    printf("lworkInBytes (prune) = %lld \n", (long long)lworkInBytes);

    if (nullptr != d_work) {
        hipFree(d_work);
    }
    cudaStat1 = hipMalloc((void**)&d_work, lworkInBytes);
    assert(hipSuccess == cudaStat1);

    /* step 4: compute csrRowPtrC and nnzC */
    status = hipsparseSpruneDense2csrNnz(  //
        handle,                           //
        m,                                //
        n,                                //
        d_A,                              //
        lda,                              //
        &threshold,                       //
        descrC,                           //
        d_csrRowPtrC,                     //
        &nnzC,                            // host
        d_work);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    cudaStat1 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);

    if (0 == nnzC) cout << log_info << "No outlier." << endl;

    /* step 5: compute csrColIndC and csrValC */
    cudaStat1 = hipMalloc((void**)&d_csrColIndC, sizeof(int) * nnzC);
    cudaStat2 = hipMalloc((void**)&d_csrValC, sizeof(float) * nnzC);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    status = hipsparseSpruneDense2csr(  //
        handle,                        //
        m,                             //
        n,                             //
        d_A,                           //
        lda,                           //
        &threshold,                    //
        descrC,                        //
        d_csrValC,                     //
        d_csrRowPtrC,                  //
        d_csrColIndC,                  //
        d_work);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    cudaStat1 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);

    /* step 6: output C */
    //    csrRowPtrC = (int*)malloc(sizeof(int) * (m + 1));
    //    csrColIndC = (int*)malloc(sizeof(int) * nnzC);
    //    csrValC    = (float*)malloc(sizeof(float) * nnzC);
    *csrRowPtrC = new int[m + 1];
    *csrColIndC = new int[nnzC];
    *csrValC    = new float[nnzC];
    assert(nullptr != csrRowPtrC);
    assert(nullptr != csrColIndC);
    assert(nullptr != csrValC);

    cudaStat1 = hipMemcpy(*csrRowPtrC, d_csrRowPtrC, sizeof(int) * (m + 1), hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(*csrColIndC, d_csrColIndC, sizeof(int) * nnzC, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(*csrValC, d_csrValC, sizeof(float) * nnzC, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    //    printCsr(m, n, nnzC, descrC, csrValC, csrRowPtrC, csrColIndC, "C");

    /* free resources */
    if (d_A) hipFree(d_A);
    if (d_csrRowPtrC) hipFree(d_csrRowPtrC);
    if (d_csrColIndC) hipFree(d_csrColIndC);
    if (d_csrValC) hipFree(d_csrValC);

    //    if (csrRowPtrC) free(csrRowPtrC);
    //    if (csrColIndC) free(csrColIndC);
    //    if (csrValC) free(csrValC);

    //    for (auto i = 0; i < 200; i++) cout << i << "\t" << csrColIndC[i] << "\t" << csrValC[i] << endl;

    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (descrC) hipsparseDestroyMatDescr(descrC);

    //    hipDeviceReset();
}