// 20-09-10

#include <bits/stdint-uintn.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cassert>
#include <iostream>
using std::cout;
using std::endl;

#include "cuda_error_handling.cuh"
#include "format.hh"
#include "gather_scatter.cuh"
#include "io.hh"

template <typename DType>
void cusz::impl::GatherAsCSR(DType* d_A, size_t lenA, size_t ldA, int* nnz, std::string* fo)
{
    // dealing with outlier
    uint8_t* outbin;
    size_t   lrp, lci, lv, ltotal;

    {
        hipsparseHandle_t   handle      = nullptr;
        hipStream_t       stream      = nullptr;
        hipsparseMatDescr_t descr       = nullptr;
        const int          m           = ldA;
        const int          n           = ldA;
        int*               d_nnzPerRow = nullptr;
        int*               d_csrRowPtr = nullptr;
        int*               d_csrColInd = nullptr;
        DType*             d_csrVal    = nullptr;

        // clang-format off
        CHECK_CUDA(hipStreamCreateWithFlags(   &stream, hipStreamNonBlocking        )); // 1. create stream
        CHECK_CUSPARSE(hipsparseCreate(          &handle                               )); // 2. create handle
        CHECK_CUSPARSE(hipsparseSetStream(        handle, stream                       )); // 3. bind stream
        CHECK_CUSPARSE(hipsparseCreateMatDescr(  &descr                                )); // 4. create descr
        CHECK_CUSPARSE(hipsparseSetMatIndexBase(  descr,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
        CHECK_CUSPARSE(hipsparseSetMatType(       descr,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // typ
        // clang-format on

        // compute nnz
        CHECK_CUDA(hipMalloc((void**)&d_nnzPerRow, sizeof(int) * m));

        CHECK_CUSPARSE(hipsparseSnnz(
            handle, HIPSPARSE_DIRECTION_ROW,  // parsed by row
            m, n, descr, d_A, ldA,           // descrption of d_A
            d_nnzPerRow, nnz)                // output
        );

        cout << "nnz: " << *nnz << endl;

        lrp    = sizeof(int) * (m + 1);
        lci    = sizeof(int) * *nnz;
        lv     = sizeof(DType) * *nnz;
        ltotal = lrp + lci + lv;
        // csrRowPtr   = new int[m + 1];
        // csrColInd   = new int[*nnz];
        // csrVal      = new DType[*nnz];
        outbin = new uint8_t[ltotal];
        CHECK_CUDA(hipMalloc((void**)&d_csrRowPtr, lrp));
        CHECK_CUDA(hipMalloc((void**)&d_csrColInd, lci));
        CHECK_CUDA(hipMalloc((void**)&d_csrVal, lv));

        CHECK_CUSPARSE(hipsparseSdense2csr(
            handle,                              //
            m, n, descr, d_A, ldA,               // descritpion of d_A
            d_nnzPerRow,                         // prefileld by nnz() func
            d_csrVal, d_csrRowPtr, d_csrColInd)  // output
        );
        CHECK_CUDA(hipDeviceSynchronize());

        // clang-format off
        // CHECK_CUDA(hipMemcpy(csrRowPtr, d_csrRowPtr, lrp, hipMemcpyDeviceToHost));
        // CHECK_CUDA(hipMemcpy(csrColInd, d_csrColInd, lci, hipMemcpyDeviceToHost));
        // CHECK_CUDA(hipMemcpy(csrVal,    d_csrVal,    lv,  hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(outbin,             d_csrRowPtr, lrp, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(outbin + lrp,       d_csrColInd, lci, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(outbin + lrp + lci, d_csrVal,    lv,  hipMemcpyDeviceToHost));
        // clang-format on

        auto csrval = reinterpret_cast<DType*>(outbin + lrp + lci);
        int  count  = 0;
        for (auto i = 0; i < *nnz; i++) {
            if (csrval[i] != 0) {
                // cout << i << "\t" << csrVal[i] << endl;
                count++;
            }
        }
        cout << "non zero count again: " << count << endl;

        // clean up
        if (d_csrRowPtr) hipFree(d_csrRowPtr);
        if (d_csrColInd) hipFree(d_csrColInd);
        if (d_csrVal) hipFree(d_csrVal);
        if (d_nnzPerRow) hipFree(d_nnzPerRow);
        if (handle) hipsparseDestroy(handle);
        if (stream) hipStreamDestroy(stream);
        if (descr) hipsparseDestroyMatDescr(descr);
    }

    cout << log_dbg << "outlier_bin byte length:\t" << ltotal << endl;

    io::WriteBinaryFile(outbin, ltotal, fo);
    cout << log_info << "Saved outlier in CSR format." << endl;
    delete[] outbin;
};

template void cusz::impl::GatherAsCSR<float>(float* d_A, size_t lenA, size_t ldA, int* nnz, std::string* fo);

template <typename DType>
void cusz::impl::ScatterFromCSR(DType* d_A, size_t lenA, size_t ldA, int* nnz, std::string* fi)
{
    // clang-format off
    auto lrp   = sizeof(int)   * (ldA + 1);
    auto lci   = sizeof(int)   * *nnz;
    auto lv      = sizeof(DType) * *nnz;
    auto l_total       = lrp + lci + lv;
    auto outlier_bin   = io::ReadBinaryFile<uint8_t>(*fi, l_total);
    auto csrRowPtr     = reinterpret_cast<int*  >(outlier_bin);
    auto csrColInd     = reinterpret_cast<int*  >(outlier_bin + lrp);
    auto csrVal        = reinterpret_cast<DType*>(outlier_bin + lrp + lci);  // TODO template
    // clang-format on

    int count = 0;
    for (auto i = 0; i < *nnz; i++) {
        if (csrVal[i] != 0) {
            // cout << i << "\t" << csrVal[i] << endl;
            count++;
        }
    }
    cout << "non zero count again (extract): " << count << endl;
    cout << log_dbg << "outlier_bin byte length:\t" << l_total << endl;

    {
        hipsparseHandle_t   handle      = nullptr;
        hipStream_t       stream      = nullptr;
        hipsparseMatDescr_t descr       = nullptr;
        const int          m           = ldA;
        const int          n           = m;
        int*               d_csrRowPtr = nullptr;
        int*               d_csrColInd = nullptr;
        DType*             d_csrVal    = nullptr;

        // clang-format off
        CHECK_CUDA(hipStreamCreateWithFlags(   &stream, hipStreamNonBlocking        )); // 1. create stream
        CHECK_CUSPARSE(hipsparseCreate(          &handle                               )); // 2. create handle
        CHECK_CUSPARSE(hipsparseSetStream(        handle, stream                       )); // 3. bind stream
        CHECK_CUSPARSE(hipsparseCreateMatDescr(  &descr                                )); // 4. create descr
        CHECK_CUSPARSE(hipsparseSetMatIndexBase(  descr,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
        CHECK_CUSPARSE(hipsparseSetMatType(       descr,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type

        CHECK_CUDA(hipMalloc( (void**)&d_csrRowPtr,   lrp ));
        CHECK_CUDA(hipMalloc( (void**)&d_csrColInd,   lci ));
        CHECK_CUDA(hipMalloc( (void**)&d_csrVal,      lv    ));
        CHECK_CUDA(hipMemcpy( d_csrRowPtr, csrRowPtr, lrp, hipMemcpyHostToDevice ));
        CHECK_CUDA(hipMemcpy( d_csrColInd, csrColInd, lci, hipMemcpyHostToDevice ));
        CHECK_CUDA(hipMemcpy( d_csrVal,    csrVal,    lv,    hipMemcpyHostToDevice ));
        // clang-format on

        // fill
        CHECK_CUSPARSE(hipsparseScsr2dense(handle, m, n, descr, d_csrVal, d_csrRowPtr, d_csrColInd, d_A, ldA));
        CHECK_CUDA(hipDeviceSynchronize());

        if (d_csrRowPtr) hipFree(d_csrRowPtr);
        if (d_csrColInd) hipFree(d_csrColInd);
        if (d_csrVal) hipFree(d_csrVal);
        if (handle) hipsparseDestroy(handle);
        if (stream) hipStreamDestroy(stream);
        if (descr) hipsparseDestroyMatDescr(descr);
    }

    cout << log_info << "Extracted outlier from CSR format." << endl;

    delete[] outlier_bin;
}

template void cusz::impl::ScatterFromCSR<float>(float* d_A, size_t lenA, size_t ldA, int* nnz, std::string* fi);

void cusz::impl::PruneGatherAsCSR(
    float*       d_A,  //
    size_t       lenA,
    const int    m,
    int&         nnzC,
    std::string* fo)
{
    hipsparseHandle_t   handle = nullptr;
    hipStream_t       stream = nullptr;
    hipsparseMatDescr_t descrC = nullptr;
    const int          lda    = m;
    const int          n      = m;  // square
    // int*               csrRowPtrC   = nullptr;
    // int*               csrColIndC   = nullptr;
    // float*             csrValC      = nullptr;
    int*   d_csrRowPtrC = nullptr;
    int*   d_csrColIndC = nullptr;
    float* d_csrValC    = nullptr;
    size_t lworkInBytes = 0;
    char*  d_work       = nullptr;

    float threshold = 0; /* remove Aij <= 4.1 */

    // clang-format off
    CHECK_CUDA(hipStreamCreateWithFlags(   &stream, hipStreamNonBlocking        )); // 1. create stream
    CHECK_CUSPARSE(hipsparseCreate(          &handle                               )); // 2. create handle
    CHECK_CUSPARSE(hipsparseSetStream(        handle, stream                       )); // 3. bind stream
    CHECK_CUSPARSE(hipsparseCreateMatDescr(  &descrC                               )); // 4. create descr
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(  descrC, HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
    CHECK_CUSPARSE(hipsparseSetMatType(       descrC, HIPSPARSE_MATRIX_TYPE_GENERAL )); // typ
    // clang-format on

    CHECK_CUDA(hipMalloc((void**)&d_csrRowPtrC, sizeof(int) * (m + 1)));

    CHECK_CUSPARSE(hipsparseSpruneDense2csr_bufferSizeExt(  //
        handle,                                            //
        m,                                                 //
        n,                                                 //
        d_A,                                               //
        lda,                                               //
        &threshold,                                        //
        descrC,                                            //
        d_csrValC,                                         //
        d_csrRowPtrC,                                      //
        d_csrColIndC,                                      //
        &lworkInBytes));

    //    printf("lworkInBytes (prune) = %lld \n", (long long)lworkInBytes);

    if (nullptr != d_work) {
        hipFree(d_work);
    }
    CHECK_CUDA(hipMalloc((void**)&d_work, lworkInBytes));

    /* step 4: compute csrRowPtrC and nnzC */
    CHECK_CUSPARSE(hipsparseSpruneDense2csrNnz(  //
        handle,                                 //
        m,                                      //
        n,                                      //
        d_A,                                    //
        lda,                                    //
        &threshold,                             //
        descrC,                                 //
        d_csrRowPtrC,                           //
        &nnzC,                                  // host
        d_work));

    CHECK_CUDA(hipDeviceSynchronize());

    if (0 == nnzC) cout << log_info << "No outlier." << endl;

    /* step 5: compute csrColIndC and csrValC */
    CHECK_CUDA(hipMalloc((void**)&d_csrColIndC, sizeof(int) * nnzC));
    CHECK_CUDA(hipMalloc((void**)&d_csrValC, sizeof(float) * nnzC));

    CHECK_CUSPARSE(hipsparseSpruneDense2csr(  //
        handle,                              //
        m,                                   //
        n,                                   //
        d_A,                                 //
        lda,                                 //
        &threshold,                          //
        descrC,                              //
        d_csrValC,                           //
        d_csrRowPtrC,                        //
        d_csrColIndC,                        //
        d_work));
    CHECK_CUDA(hipDeviceSynchronize());

    /* step 6: output C */
    auto lrp    = sizeof(int) * (m + 1);
    auto lci    = sizeof(int) * nnzC;
    auto lv     = sizeof(float) * nnzC;
    auto ltotal = lrp + lci + lv;
    auto outbin = new uint8_t[ltotal];

    // CHECK_CUDA(hipMemcpy(csrRowPtrC, d_csrRowPtrC, sizeof(int) * (m + 1), hipMemcpyDeviceToHost));
    // CHECK_CUDA(hipMemcpy(csrColIndC, d_csrColIndC, sizeof(int) * nnzC, hipMemcpyDeviceToHost));
    // CHECK_CUDA(hipMemcpy(csrValC, d_csrValC, sizeof(float) * nnzC, hipMemcpyDeviceToHost));
    // clang-format off
    CHECK_CUDA(hipMemcpy(outbin,             d_csrRowPtrC, lrp, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(outbin + lrp,       d_csrColIndC, lci, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(outbin + lrp + lci, d_csrValC,    lv,  hipMemcpyDeviceToHost));
    // clang-format on

    io::WriteBinaryFile(outbin, ltotal, fo);

    // printCsr(m, n, nnzC, descrC, csrValC, csrRowPtrC, csrColIndC, "C");

    /* free resources */
    if (d_A) hipFree(d_A);
    if (d_csrRowPtrC) hipFree(d_csrRowPtrC);
    if (d_csrColIndC) hipFree(d_csrColIndC);
    if (d_csrValC) hipFree(d_csrValC);

    if (outbin) delete[] outbin;

    //    for (auto i = 0; i < 200; i++) cout << i << "\t" << csrColIndC[i] << "\t" << csrValC[i] << endl;

    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (descrC) hipsparseDestroyMatDescr(descrC);
}
